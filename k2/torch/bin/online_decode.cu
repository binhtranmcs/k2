#include "hip/hip_runtime.h"
/**
 * Copyright      2021  Xiaomi Corporation (authors: Fangjun Kuang, Wei Kang)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */
#include <cassert>
#include <cstdio>
#include <cstdlib>
#include <string>
#include <vector>
#include <dirent.h>

#include "k2/csrc/intersect_dense_pruned.h"
#include "k2/torch/csrc/decode.h"
#include "k2/torch/csrc/dense_fsa_vec.h"
#include "k2/torch/csrc/deserialization.h"
#include "k2/torch/csrc/features.h"
#include "k2/torch/csrc/fsa_algo.h"
#include "k2/torch/csrc/symbol_table.h"
#include "k2/torch/csrc/utils.h"
#include "k2/torch/csrc/wave_reader.h"
#include "kaldifeat/csrc/feature-fbank.h"
#include "torch/all.h"
#include "torch/script.h"
#include "torch/utils.h"

C10_DEFINE_bool(use_gpu, false, "True to use GPU. False to use CPU");
C10_DEFINE_string(jit_pt, "", "Path to exported jit file.");
C10_DEFINE_bool(use_ctc_decoding, true, "True to use CTC decoding");
C10_DEFINE_string(hlg, "/home/cpu13266/binhtt4/clone/k2/build/hlg/my_HLG.pt",
                  "Path to HLG.pt. Needed if --use_ctc_decoding is false");
C10_DEFINE_string(word_table, "/home/cpu13266/binhtt4/clone/k2/build/hlg/my_words.txt",
                  "Path to words.txt. Needed if --use_ctc_decoding is false");
C10_DEFINE_string(tokens, "",
                  "Path to a tokens.txt. Needed if --use_ctc_decoding is true");
// Fsa decoding related
C10_DEFINE_double(search_beam, 20, "search_beam in IntersectDensePruned");
C10_DEFINE_double(output_beam, 20, "output_beam in IntersectDensePruned");
C10_DEFINE_int(min_activate_states, 30,
               "min_activate_states in IntersectDensePruned");
C10_DEFINE_int(max_activate_states, 10000,
               "max_activate_states in IntersectDensePruned");
// fbank related
C10_DEFINE_int(sample_rate, 16000, "Expected sample rate of wave files");
C10_DEFINE_double(frame_shift_ms, 10.0,
                  "Frame shift in ms for computing Fbank");
C10_DEFINE_double(frame_length_ms, 25.0,
                  "Frame length in ms for computing Fbank");
C10_DEFINE_int(num_bins, 80, "Number of triangular bins for computing Fbank");
C10_DEFINE_int(num_streams, 1, "Number of concurrent streams");

C10_DEFINE_string(audio_dir,
                  "/home/cpu13266/binhtt4/asr/e2e/asr-end2end-service/build/btt4_audios/",
                  "audio directory for testing");

static void CheckArgs() {
#if !defined(K2_WITH_CUDA)
  if (FLAGS_use_gpu) {
    std::cerr << "k2 was not compiled with CUDA"
              << "\n";
    std::cerr << "Please use --use_gpu 0"
              << "\n";
    exit(EXIT_FAILURE);
  }
#endif

  if (FLAGS_jit_pt.empty()) {
    std::cerr << "Please provide --jit_pt"
              << "\n";
    std::cerr << torch::UsageMessage() << "\n";
    exit(EXIT_FAILURE);
  }

  if (FLAGS_use_ctc_decoding && FLAGS_tokens.empty()) {
    std::cout << "Please provide --tokens"
              << "\n";
    std::cout << torch::UsageMessage() << "\n";
    exit(EXIT_FAILURE);
  }

  if (FLAGS_use_ctc_decoding == false && FLAGS_hlg.empty()) {
    std::cerr << "Please provide --hlg"
              << "\n";
    std::cerr << torch::UsageMessage() << "\n";
    exit(EXIT_FAILURE);
  }

  if (FLAGS_use_ctc_decoding == false && FLAGS_word_table.empty()) {
    std::cerr << "Please provide --word_table"
              << "\n";
    std::cerr << torch::UsageMessage() << "\n";
    exit(EXIT_FAILURE);
  }
}


std::vector<char> get_the_bytes(std::string filename) {
  std::ifstream input(filename, std::ios::binary);
  std::vector<char> bytes(
      (std::istreambuf_iterator<char>(input)),
      (std::istreambuf_iterator<char>()));

  input.close();
  return bytes;
}


int main(int argc, char *argv[]) {
  // see
  // https://pytorch.org/docs/stable/notes/cpu_threading_torchscript_inference.html
  torch::set_num_threads(1);
  torch::set_num_interop_threads(1);

  std::string usage = R"(
  (1) CTC decoding
    ./bin/online_decode \
      --use_ctc_decoding true \
      --jit_pt <path to exported torch script pt file> \
      --tokens <path to tokens.txt> \
      /path/to/foo.wav \
      /path/to/bar.wav \
      <more wave files if any>
  (2) HLG decoding
    ./bin/online_decode \
      --use_ctc_decoding false \
      --jit_pt <path to exported torch script pt file> \
      --hlg <path to HLG.pt> \
      --word_table <path to words.txt> \
      /path/to/foo.wav \
      /path/to/bar.wav \
      <more wave files if any>
   --use_gpu false to use CPU
   --use_gpu true to use GPU
  )";
  torch::SetUsageMessage(usage);

  torch::ParseCommandLineFlags(&argc, &argv);
  CheckArgs();

  torch::Device device(torch::kCPU);
  if (FLAGS_use_gpu) {
    device = torch::Device(torch::kCUDA, 0);
  }

  K2_LOG(INFO) << "Device: " << device;

  int32_t num_waves = 1;

  std::ofstream fout("fixk2.txt");
  std::vector<int64_t> num_frames;

  /// feat
  kaldifeat::FbankOptions fbank_opts;
  fbank_opts.frame_opts.samp_freq = FLAGS_sample_rate;
  fbank_opts.frame_opts.dither = 0;
  fbank_opts.frame_opts.frame_shift_ms = FLAGS_frame_shift_ms;
  fbank_opts.frame_opts.frame_length_ms = FLAGS_frame_length_ms;
  fbank_opts.mel_opts.num_bins = FLAGS_num_bins;
  fbank_opts.device = device;
  kaldifeat::Fbank fbank(fbank_opts);

  /// jit
  K2_LOG(INFO) << "Load neural network model";
  torch::jit::script::Module module = torch::jit::load(FLAGS_jit_pt);
  module.eval();
  module.to(device);

  /// fst
  K2_LOG(INFO) << "Load " << FLAGS_hlg;
  k2::FsaClass decoding_graph = k2::LoadFsa(FLAGS_hlg, device);
  K2_CHECK(decoding_graph.HasTensorAttr("aux_labels") ||
           decoding_graph.HasRaggedTensorAttr("aux_labels"));
  decoding_graph.SetTensorAttr("lm_scores", decoding_graph.Scores().clone());
  auto decoding_fsa = k2::FsaToFsaVec(decoding_graph.fsa);
  k2::OnlineDenseIntersecter decoder(
      decoding_fsa, FLAGS_num_streams, FLAGS_search_beam, FLAGS_output_beam,
      FLAGS_min_activate_states, FLAGS_max_activate_states);

  /// simulation
  DIR *dir; struct dirent *diread;
  std::string path = FLAGS_audio_dir;
//  std::string path = "/home/cpu13266/Downloads/test/";
//  std::string path = "/home/cpu13266/binhtt4/clone/k2/build/tmp/tmpwav/";
  if ((dir = opendir(path.c_str())) == nullptr) {
    assert(false);
  }
  while ((diread = readdir(dir)) != nullptr) {
//  while (true) {
    std::string filename = path + diread->d_name;
    if (filename.find(".wav") == std::string::npos) continue;

    std::cout << diread->d_name << '\n';
    fout << diread->d_name << ' ';

    K2_LOG(INFO) << "Load wave files " << diread->d_name;
    auto wave_data = k2::ReadWave(std::vector<std::string>(1, filename), FLAGS_sample_rate);
    for (auto &w : wave_data) w = w.to(device);

    {
      /// feat
      K2_LOG(INFO) << "Compute features";
      std::vector<int64_t> num_frames;
      auto features_vec = k2::ComputeFeatures(fbank, wave_data, &num_frames);
      // Note: math.log(1e-10) is -23.025850929940457
      auto features = torch::nn::utils::rnn::pad_sequence(features_vec, true,
          -23.025850929940457f);

      /// jit
      int32_t subsampling_factor = module.attr("subsampling_factor").toInt();
      torch::Dict<std::string, torch::Tensor> sup;
      sup.insert("sequence_idx", torch::arange(num_waves, torch::kInt));
      sup.insert("start_frame", torch::zeros({num_waves}, torch::kInt));
      sup.insert("num_frames",
          torch::from_blob(num_frames.data(), {num_waves}, torch::kLong)
              .to(torch::kInt));
      torch::IValue supervisions(sup);
      std::vector<torch::IValue> inputs;
      inputs.emplace_back(std::move(features));
      inputs.emplace_back(supervisions);
      K2_LOG(INFO) << "Compute nnet_output";
      // the output for module.forward() is a tuple of 3 tensors
      auto outputs = module.forward(inputs).toTuple();
      assert(outputs->elements().size() == 3u);
      auto nnet_output = outputs->elements()[0].toTensor();

//      int subsampling_factor = 4;
//      num_frames.push_back(96 * 4);
//      std::vector<char> f = get_the_bytes("/home/cpu13266/binhtt4/clone/k2/build/tmp.pt");
//      torch::IValue x = torch::pickle_load(f);
//      torch::Tensor nnet_output = x.toTensor();

      K2_LOG(INFO) << "Sizes: " << nnet_output.sizes() << " | " << num_frames[0];

      /// fst
      // store decode states for each waves
      std::vector<k2::DecodeStateInfo> states_info(num_waves);
      // decocding results for each waves
      std::vector<std::string> texts(num_waves, "");
      std::vector<int32_t> positions(num_waves, 0);
      int32_t T = nnet_output.size(1);
      int32_t chunk_size = 10;

      /// simulate asynchronous decoding
      while (true) {
        k2::DecodeStateInfo dummy_state_info;
        std::vector<k2::DecodeStateInfo*> current_states_info;
        std::vector<int64_t> num_frame;
        std::vector<torch::Tensor> current_nnet_output;
        // which waves we are decoding now
        std::vector<int32_t> current_wave_ids;

        {
          for (int32_t i = 0; i < num_waves; ++i) {
            // this wave is done
            if (num_frames[i] == 0) continue;

            current_states_info.push_back(&states_info[i]);
            current_wave_ids.push_back(i);

            if (num_frames[i] <= chunk_size * subsampling_factor) {
              num_frame.push_back(num_frames[i]);
              num_frames[i] = 0;
            } else {
              num_frame.push_back(chunk_size * subsampling_factor);
              num_frames[i] -= chunk_size * subsampling_factor;
            }

            int32_t start = positions[i],
                    end = start + chunk_size >= T ? T : start + chunk_size;
            positions[i] = end;
            auto sub_output = nnet_output.index(
                {i, torch::indexing::Slice(start, end), torch::indexing::Slice()});

            // padding T axis to chunk_size if needed
            namespace F = torch::nn::functional;
            sub_output = F::pad(sub_output,
                                F::PadFuncOptions({0, 0, 0, chunk_size - end + start})
                                    .mode(torch::kConstant));

            current_nnet_output.push_back(sub_output);

            // we can only decode `FLAGS_num_streams` waves at a time
            if (static_cast<int32_t>(current_wave_ids.size()) >= FLAGS_num_streams)
              break;
          }
          if (current_wave_ids.size() == 0) break;  // finished
          // no enough waves, feed in garbage data
          while (static_cast<int32_t>(num_frame.size()) < FLAGS_num_streams) {
            assert(false);
            num_frame.push_back(0);
            auto opts = torch::TensorOptions().dtype(nnet_output.dtype())
                            .device(nnet_output.device());
            current_nnet_output.push_back(
                torch::zeros({chunk_size, nnet_output.size(2)}, opts));
            current_states_info.push_back(&dummy_state_info);
          }
        }

        /// sup
        auto sub_nnet_output = torch::stack(current_nnet_output);
        torch::Dict<std::string, torch::Tensor> sup;
        sup.insert("sequence_idx", torch::arange(FLAGS_num_streams, torch::kInt));
        sup.insert("start_frame", torch::zeros({FLAGS_num_streams}, torch::kInt));
        sup.insert("num_frames",
            torch::from_blob(num_frame.data(), {FLAGS_num_streams},
                torch::kLong).to(torch::kInt));
        torch::IValue supervision(sup);
        torch::Tensor supervision_segments =
            k2::GetSupervisionSegments(supervision, subsampling_factor);

        /// get lattice
        K2_LOG(INFO) << "Decoding";
        k2::DenseFsaVec dense_fsa_vec = k2::CreateDenseFsaVec(
            sub_nnet_output, supervision_segments, subsampling_factor - 1);
        k2::FsaVec fsa;
        k2::Array1<int32_t> graph_arc_map;
        decoder.Decode(dense_fsa_vec, &current_states_info, &fsa, &graph_arc_map);
        k2::FsaClass lattice(fsa);
        lattice.CopyAttrs(decoding_graph, k2::Array1ToTorch<int32_t>(graph_arc_map));

        /// nbest
        // lattice = k2::ShortestPath(lattice);
        auto nbest = k2::Nbest::FromLattice(lattice, 3, 1.0);
        nbest.Intersect(&lattice);
        // must hold the array first before getting the underlying data
        auto am_scores_arr = k2::Array1FromTorch<float>(nbest.ComputeAmScores()).To(k2::GetCpuContext());
        auto lm_scores_arr = k2::Array1FromTorch<float>(nbest.ComputeLmScores()).To(k2::GetCpuContext());
        auto row_split = nbest.shape.RowSplits(1).To(k2::GetCpuContext());

        auto ragged_aux_labels = k2::GetTexts(nbest.fsa);
        K2_LOG(INFO) << ragged_aux_labels.shape;
        auto aux_labels_vec = ragged_aux_labels.ToVecVec();
      }
    }
  }

  return 0;
}
