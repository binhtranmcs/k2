#include "hip/hip_runtime.h"
/**
 * Copyright      2020  Mobvoi Inc.        (authors: Fangjun Kuang)
 *                      Xiaomi Corporation (authors: Haowen Qiu)
 *
 * See LICENSE for clarification regarding multiple authors
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <cstdlib>
#include <mutex>  // NOLINT

#include "k2/csrc/context.h"
#include "k2/csrc/log.h"
#include "k2/csrc/nvtx.h"
#include "k2/csrc/pytorch_context.h"

namespace k2 {

static constexpr std::size_t kAlignment = 64;

// TODO(haowen): most of implementations below should be updated later.
class CpuContext : public Context {
 public:
  CpuContext() = default;
//  ContextPtr GetCpuContext() override { return shared_from_this(); }
  DeviceType GetDeviceType() const override { return kCpu; }

  void *Allocate(std::size_t bytes, void **deleter_context) override {
    void *p = nullptr;
    if (bytes) {
      int32_t ret = posix_memalign(&p, kAlignment, bytes);
      K2_CHECK_EQ(ret, 0);
    }
    if (deleter_context != nullptr) *deleter_context = nullptr;
    return p;
  }

  bool IsCompatible(const Context &other) const override {
    return other.GetDeviceType() == kCpu;
  }

  void Deallocate(void *data, void * deleter_context) override {
      if (deleter_context != nullptr) {
          // a non-empty `deleter_context` indicates that
          // the memory is passed from a `torch::Tensor`
          delete reinterpret_cast<ManagedTensor *>(deleter_context);
      } else {
          free(data);
      }
  }

  void CopyDataTo(size_t num_bytes, const void *src,
                  ContextPtr dst_context, void *dst) override {
      memcpy(dst, src, num_bytes);
  };

};

class CudaContext : public Context {
 public:
  explicit CudaContext(int32_t gpu_id) : gpu_id_(gpu_id) {
#ifdef K2_WITH_CUDA
    if (gpu_id_ != -1) {
      auto ret = hipSetDevice(gpu_id_);
      K2_CHECK_CUDA_ERROR(ret);
    }
    // TODO(haowen): choose one from available GPUs if gpu_id == -1?
    // and handle GPU ids from multiple machines.
    auto ret = hipStreamCreate(&stream_);
    K2_CHECK_CUDA_ERROR(ret);
#endif
  }
//  ContextPtr GetCpuContext() override { return k2::GetCpuContext(); }
  DeviceType GetDeviceType() const override { return kCuda; }
  int32_t GetDeviceId() const override { return gpu_id_; }

  void *Allocate(std::size_t bytes, void **deleter_context) override {
    void *p = nullptr;
#ifdef K2_WITH_CUDA
    if (bytes) {
      auto ret = hipMalloc(&p, bytes);
      K2_CHECK_CUDA_ERROR(ret);
    }
    if (deleter_context != nullptr) *deleter_context = nullptr;
#endif
    return p;
  }

    void CopyDataTo(size_t num_bytes, const void *src,
                    ContextPtr dst_context, void *dst) override {};

  bool IsCompatible(const Context &other) const override {
    return other.GetDeviceType() == kCuda && other.GetDeviceId() == gpu_id_;
  }

  void Deallocate(void *data, void * /*deleter_context*/) override {
      #ifdef K2_WITH_CUDA
    auto ret = hipFree(data);
    K2_CHECK_CUDA_ERROR(ret);
      #endif
  }

  hipStream_t GetCudaStream() const override {
      #ifdef K2_WITH_CUDA
    return g_stream_override.OverrideStream(stream_);
      #else
      return hipStream_t{};
      #endif
  }

  void Sync() const override {
      #ifdef K2_WITH_CUDA
    auto ret = hipStreamSynchronize(stream_);
    K2_CHECK_CUDA_ERROR(ret);
      #endif
  }

  ~CudaContext() {
      #ifdef K2_WITH_CUDA
    auto ret = hipStreamDestroy(stream_);
    K2_CHECK_CUDA_ERROR(ret);
      #endif
  }

 private:
  int32_t gpu_id_;
  hipStream_t stream_;
};

ContextPtr GetCpuContext() { return std::make_shared<CpuContext>(); }

ContextPtr GetCudaContext(int32_t gpu_id /*= -1*/) {
#ifdef K2_WITH_CUDA
  static std::once_flag has_cuda_init_flag;
  static bool has_cuda = false;
  std::call_once(has_cuda_init_flag, []() {
    int n = 0;
    auto ret = hipGetDeviceCount(&n);
    if (ret == hipSuccess && n > 0)
      has_cuda = true;
    else
      K2_LOG(WARNING) << "CUDA is not available. Return a CPU context.";
  });

  if (has_cuda) return std::make_shared<CudaContext>(gpu_id);
#endif
  return GetCpuContext();
}

RegionPtr NewRegion(torch::Tensor tensor) {
    auto ans = std::make_shared<Region>();
    if (tensor.device().type() == torch::kCPU) {
        ans->context = GetCpuContext();
    } else if (tensor.is_cuda()) {
        ans->context = GetCudaContext(tensor.device().index());
    } else {
        K2_LOG(FATAL) << "Unsupported device: " << tensor.device()
                      << "\nOnly CPU and CUDA are supported";
    }

    // NOTE: the tensor is passed from Python and we have
    // to retain it to avoid potential segmentation fault.
    //
    // It will be freed in `Context::Deallocate`.
    auto *managed_tensor = new ManagedTensor(tensor);
    ans->data = tensor.data_ptr();
    ans->deleter_context = managed_tensor;
#if K2_TORCH_VERSION_MAJOR > 1 || \
(K2_TORCH_VERSION_MAJOR == 1 && K2_TORCH_VERSION_MINOR > 5)
    // nbytes() is available only for torch > 1.5
// see https://github.com/pytorch/pytorch/pull/37028
ans->num_bytes = tensor.storage().nbytes();
#else
    // capacity() is available only for torch <= 1.5.0
    ans->num_bytes = tensor.storage().capacity();
#endif
    ans->bytes_used = ans->num_bytes;
    return ans;
}

}  // namespace k2
